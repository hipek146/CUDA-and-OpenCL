#include "hip/hip_runtime.h"
// example of using CUDA streams

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <chrono>
using namespace std::chrono;

__global__
void initWith(float num, float *a, int N)
{

  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    a[i] = num;
  }
}

void initWithNoStream(float num, float *a, int N)
{

  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *vector, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(vector[i] != target)
    {
      printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
      exit(1);
    }
  }
  printf("Success! All values calculated correctly.\n");
}

int main(int argc, char** argv)
{
  int deviceId;
  int numberOfSMs;

  hipGetDevice(&deviceId);
  hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

  char* pEnd;
  const int N = 2<<strtol(argv[1], &pEnd, 10);
  //const int N = 2<<24;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;
  float *d;
  float *e;
  float *f;
  float *g;



  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);
  hipMallocManaged(&d, size);
  hipMallocManaged(&e, size);
  hipMallocManaged(&f, size);
  hipMallocManaged(&g, size);

  hipMemPrefetchAsync(a, size, deviceId);
  hipMemPrefetchAsync(b, size, deviceId);
  hipMemPrefetchAsync(c, size, deviceId);
  hipMemPrefetchAsync(d, size, deviceId);
  hipMemPrefetchAsync(e, size, deviceId);
  hipMemPrefetchAsync(f, size, deviceId);
  hipMemPrefetchAsync(g, size, deviceId);

  size_t threadsPerBlock;
  size_t numberOfBlocks;

  threadsPerBlock = 256;
  numberOfBlocks = 32 * numberOfSMs;

  hipError_t addVectorsErr;
  hipError_t asyncErr;

  /*
   * Create 3 streams to run initialize the 3 data vectors in parallel.
   */

  //auto start = high_resolution_clock::now();
  hipStream_t stream1, stream2, stream3, stream4, stream5, stream6, stream7;
  hipStreamCreate(&stream1);
  hipStreamCreate(&stream2);
  hipStreamCreate(&stream3);
  hipStreamCreate(&stream4);
  hipStreamCreate(&stream5);
  hipStreamCreate(&stream6);
  hipStreamCreate(&stream7);

  /*
   * Give each `initWith` launch its own non-standard stream.
   */

  int which = strtol(argv[2], &pEnd, 10);
  if (which == 0) {
	  initWithNoStream(1.25, a, N);
	  initWithNoStream(1.25, b, N);
	  initWithNoStream(0, c, N);
	  initWithNoStream(0, d, N);
	  initWithNoStream(1.25, e, N);
	  initWithNoStream(0, f, N);
	  initWithNoStream(1.25, g, N);
  }
  if (which == 1) {
	  initWith<<<numberOfBlocks, threadsPerBlock, 0, stream1>>>(3.1, a, N);
	  initWithNoStream(4.25, b, N);
	  initWithNoStream(0, c, N);
  }
  if (which == 2) {
	  initWith<<<numberOfBlocks, threadsPerBlock, 0, stream1>>>(3.1, a, N);
	  initWith<<<numberOfBlocks, threadsPerBlock, 0, stream2>>>(4.25, b, N);
	  initWithNoStream(0, c, N);
  }
  if (which == 3) {
	  initWithNoStream(3.1, a, N);
	  initWithNoStream(4.25, b, N);
	  initWith<<<numberOfBlocks, threadsPerBlock, 0, stream3>>>(0, c, N);
  }
  if (which == 4) {
	  initWith<<<numberOfBlocks, threadsPerBlock, 0, stream1>>>(1.25, a, N);
	  initWith<<<numberOfBlocks, threadsPerBlock, 0, stream2>>>(1.25, b, N);
	  initWith<<<numberOfBlocks, threadsPerBlock, 0, stream3>>>(0, c, N);
	  initWith<<<numberOfBlocks, threadsPerBlock, 0, stream4>>>(0, d, N);
	  initWith<<<numberOfBlocks, threadsPerBlock, 0, stream5>>>(1.25, e, N);
	  initWith<<<numberOfBlocks, threadsPerBlock, 0, stream6>>>(0, f, N);
	  initWith<<<numberOfBlocks, threadsPerBlock, 0, stream7>>>(1.25, g, N);
  }
 
  auto start = high_resolution_clock::now();
  addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);
  addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(d, c, e, N);
  addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(f, d, g, N);
  auto stop = high_resolution_clock::now();
  auto duration = duration_cast<microseconds>(stop - start);
  std::cout<< "Time in seconds: " << duration.count()/1E6 << std::endl;

  addVectorsErr = hipGetLastError();
  if(addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

  asyncErr = hipDeviceSynchronize();
  if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

  hipMemPrefetchAsync(f, size, hipCpuDeviceId);

  checkElementsAre(5, f, N);

  /*
   * Destroy streams when they are no longer needed.
   */

  hipStreamDestroy(stream1);
  hipStreamDestroy(stream2);
  hipStreamDestroy(stream3);
  hipStreamDestroy(stream4);
  hipStreamDestroy(stream5);
  hipStreamDestroy(stream6);
  hipStreamDestroy(stream7);

  hipFree(a);
  hipFree(b);
  hipFree(c);
  hipFree(d);
  hipFree(e);
  hipFree(f);
  hipFree(g);
}

