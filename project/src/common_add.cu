#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "common_header.h"
#include "util_header.h"

__global__
void addition(float *a, float *b, float* c, int N) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for(int i = index; i < N; i += stride) {
		c[i] = a[i] + b[i];
	}
}

void check_addition(float *a, float *b, float *c, int N) {
	for(int i = 0; i < N; i++) {
		if(a[i] + b[i] - c[i] > 1E-5) {
			printf("Addition result not correct\n");
			return;
		}
	}
	printf("Test Passed\n");
}


float* execute_addition(float* a, float* b, int N, size_t threads, size_t blocks, int deviceId) {
	size_t size = N * sizeof(float);

	float *c = NULL;
	checkCudaErrorState(hipMallocManaged(&c, size), "Couldn't alloc result vector for addition");

	checkCudaErrorState(hipMemPrefetchAsync(c, size, deviceId), "Couldn't send for result vector to device");

	hipStream_t vecCStream;
  	checkCudaErrorState(hipStreamCreate(&vecCStream), "Couldn't create stream for result vector");
	
	providednumberinitwithstreams<<<threads, blocks, 0, vecCStream>>>(c, N, 0);
	checkCudaErrorState(hipGetLastError(), "Problem while initializing vector c with init values for addition");
	
	addition<<<threads, blocks>>>(a, b, c, N);
	checkCudaErrorState(hipGetLastError(), "Problem adding values into result");
	
	checkCudaErrorState(hipDeviceSynchronize(), "Problem while completing device threads");

	hipStreamDestroy(vecCStream);

	check_addition(a, b, c, N);

	return c;
}
